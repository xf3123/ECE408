#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_WIDTH 3
#define BLOCK_WIDTH (TILE_WIDTH + 2)

//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int x_o = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int y_o = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int z_o = blockIdx.z * TILE_WIDTH + threadIdx.z;

  int x_i = x_o - 1;
  int y_i = y_o - 1;
  int z_i = z_o - 1;

  __shared__ float N[BLOCK_WIDTH][BLOCK_WIDTH][BLOCK_WIDTH];

  if ((x_i >= 0 && x_i < x_size) && (y_i >= 0 && y_i < y_size) && (z_i >= 0 && z_i < z_size))
  {
    N[threadIdx.z][threadIdx.y][threadIdx.x] = input[(z_i * (y_size * x_size)) + (y_i * x_size) + x_i];
  }
  else
  {
    N[threadIdx.z][threadIdx.y][threadIdx.x] = 0.0f;
  }

  __syncthreads();

  float value = 0.0f;

  if (threadIdx.x < TILE_WIDTH && threadIdx.y < TILE_WIDTH && threadIdx.z < TILE_WIDTH)
  {
    for (int i = 0; i < MASK_WIDTH; i++)
    {
      for (int j = 0; j < MASK_WIDTH; j++)
      {
        for (int k = 0; k < MASK_WIDTH; k++)
        {
          value += M[k][j][i] * N[threadIdx.z + k][threadIdx.y + j][threadIdx.x + i]; 
        }
      }
    }

    if (x_o < x_size && y_o < y_size && z_o < z_size)
    {
      output[(z_o * (y_size * x_size)) + (y_o * x_size) + x_o] = value;
    }

  }

  __syncthreads();

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void**) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void**) &deviceOutput, (inputLength - 3) * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, (hostInput + 3), (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, kernelLength * sizeof(float));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimGrid(ceil(x_size/(TILE_WIDTH * 1.0)), ceil(y_size/(TILE_WIDTH * 1.0)), ceil(z_size/(TILE_WIDTH * 1.0)));
  dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);
  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)

  hipMemcpy((hostOutput + 3), deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");
  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
